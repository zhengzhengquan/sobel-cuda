#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define CHECK(e) { int res = (e); if (res) printf("CUDA ERROR %d\n", res); }

// Used for debugging so that the output is all white
// #define THRESH -1

#define THRESH 10000

#define WARP_SIZE 32

// Both the x and y dimention of blocks as they are square
#define BLOCK_DIM 16

texture<unsigned char, 2> imageTex;

struct Image {
    int width;
    int height;
    unsigned char *img;
    unsigned char *dev_img;
};

    __global__
void filter(unsigned char *filtered, int width, int height)
{
    __shared__ int cache[BLOCK_DIM * BLOCK_DIM];

    int stride = 0;

    int tid = (blockIdx.x * gridDim.y * blockDim.x * blockDim.y) + (blockIdx.y * blockDim.x * blockDim.y) + (threadIdx.x * blockDim.y) + threadIdx.y;
    int localId = threadIdx.x * blockDim.x + threadIdx.y;

    int i = (tid + stride) % width;
    int j = (tid + stride) / width;

    while (i < width - 1 && j < height - 1)
    {
        int gradX = tex2D(imageTex, i-1, j+1) - tex2D(imageTex, i-1, j-1) + 2*tex2D(imageTex, i, j+1) - 2*tex2D(imageTex, i, j-1) + tex2D(imageTex, i+1, j+1) - tex2D(imageTex, i+1, j-1);
        int gradY = tex2D(imageTex, i-1, j-1) + 2*tex2D(imageTex, i-1, j) + tex2D(imageTex, i-1, j+1) - tex2D(imageTex, i+1, j-1) - 2*tex2D(imageTex, i+1, j) - tex2D(imageTex, i+1, j+1);

        int magnitude = (gradX * gradX) + (gradY * gradY);

        // The check for the edge pixels on the top and left boundary is made here
        // and not in the loop condition because otherwise all threads on either edges will not stride
        if (magnitude  > THRESH && i > 0 && j > 0)
        {
            cache[localId] = 255;
        }
        else
        {
            cache[localId] = 0;
        }

        __syncthreads();

        filtered[j * width + i] = cache[localId];

        stride += gridDim.x * gridDim.y * blockDim.x * blockDim.y;

        i = (tid + stride) % width;
        j = (tid + stride) / width;
    }
}

// Save an image to file
void saveImage(char*, Image*);

// Read PBM image from file
Image readImage(char*);

// Convert an image to Gray Scale
Image convertGrayScale(Image*);

// Run the sobel image filter
Image runFilter(Image*);

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        printf("Usage: exec filename filename\n");
        exit(1);
    }

    char *fname = argv[1];
    char *fname2 = argv[2];

    // Read Original Image
    Image source = readImage(fname);

    // Convert to Gray Scale
    Image grayScale = convertGrayScale(&source);

    // Filter the image
    Image filtered = runFilter(&grayScale);

    // Save back to a file
    saveImage(fname2, &filtered);

    // Do civil duty and free memory
    free(source.img);
    free(grayScale.img);
    free(filtered.img);

    exit(0);
}

Image runFilter(Image *grayScale)
{
    // Creating a new black Image
    int pixels = grayScale->width * grayScale->height;
    int imageSize = grayScale->width * grayScale->height * sizeof(unsigned char);

    Image filtered;
    filtered.width = grayScale->width;
    filtered.height = grayScale->height;
    filtered.img = (unsigned char *)malloc(pixels);

    unsigned char *devGrayScale;
    unsigned char *devFiltered;

    // Initialize Cuda Memory
    CHECK(hipMalloc(&devGrayScale, imageSize));
    CHECK(hipMalloc(&devFiltered, imageSize));

    // Copy and Initialize Cuda Memory
    CHECK(hipMemcpy(devGrayScale, grayScale->img, imageSize, hipMemcpyHostToDevice));
    CHECK(hipMemset(devFiltered, 0, imageSize));

    // Initalize texture
    hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
    CHECK(hipBindTexture2D(NULL, imageTex, devGrayScale, desc, grayScale->width, grayScale->height, sizeof(unsigned char) * grayScale->width));

    // Initialize Stopwatch
    hipEvent_t event1, event2;
    hipEventCreate(&event1);
    hipEventCreate(&event2);
    hipEventRecord(event1, 0);

    // Run the kernel
    // The y dimention is set 1 only to demonstrate the stride!
    dim3 dimBlock(grayScale->width/BLOCK_DIM, 1);
    dim3 dimGrid(BLOCK_DIM, BLOCK_DIM);
    filter<<<dimBlock, dimGrid>>>(devFiltered, filtered.width, filtered.height);

    // Stop Stopwatch
    hipEventRecord(event2, 0);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
    float dt_ms = 0;
    hipEventElapsedTime(&dt_ms, event1, event2);
    printf("The filter ran in : %f seconds.\n", dt_ms);

    // Return the Cuda Memory
    CHECK(hipMemcpy(filtered.img, devFiltered, imageSize, hipMemcpyDeviceToHost));

    // Free Cuda Texture and Memory
    CHECK(hipUnbindTexture(imageTex));
    CHECK(hipFree(devGrayScale));
    CHECK(hipFree(devFiltered));

    return filtered;
}

Image readImage(char *fname)
{
    Image source;

    FILE *src;

    if (!(src = fopen(fname, "rb")))
    {
        printf("Couldn't open file %s for reading.\n", fname);
        exit(1);
    }

    char p,s;
    fscanf(src, "%c%c\n", &p, &s);
    if (p != 'P' || s != '6')
    {
        printf("Not a valid PPM file (%c %c)\n", p, s);
        exit(1);
    }

    fscanf(src, "%d %d\n", &source.width, &source.height);
    int ignored;
    fscanf(src, "%d\n", &ignored);

    int pixels = source.width * source.height;
    source.img = (unsigned char *)malloc(pixels*3);
    if (fread(source.img, sizeof(unsigned char), pixels*3, src) != pixels*3)
    {
        printf("Error reading file.\n");
        exit(1);
    }
    fclose(src);

    return source;
}

void saveImage(char *fname, Image *source)
{
    int pixels = source->width * source->height;

    FILE *out;

    if (!(out = fopen(fname, "wb")))
    {
        printf("Couldn't open file for output.\n");
        exit(1);
    }

    fprintf(out, "P5\n%d %d\n255\n", source->width, source->height);

    if (fwrite(source->img, sizeof(unsigned char), pixels, out) != pixels)
    {
        printf("Error writing file.\n");
        exit(1);
    }

    fclose(out);
}

Image convertGrayScale(Image *source)
{
    int pixels = source->width * source->height;

    Image grayScale;
    grayScale.width = source->width;
    grayScale.height = source->height;
    grayScale.img = (unsigned char *)malloc(pixels);
    for (int i = 0; i < pixels; i++)
    {
        unsigned int r = source->img[i*3];
        unsigned int g = source->img[i*3 + 1];
        unsigned int b = source->img[i*3 + 2];
        grayScale.img[i] = 0.2989*r + 0.5870*g + 0.1140*b;
    }

    return grayScale;
}
